
#include <hip/hip_runtime.h>
__device__ void decipher(unsigned int, unsigned int*, unsigned int const*);


__global__ void decrypt_bytes(unsigned int *decrypted, unsigned char *key, unsigned int *encrypted)
{
    const int tx = threadIdx.x + (blockIdx.x * blockDim.x);
    //finne ut hvordan man deler de opp paa traadene 
    unsigned int deciphered[2];
    deciphered[0] = encrypted[0];
    deciphered[1] = encrypted[1];
    decipher(32, deciphered, (unsigned int*)key);
    decrypted[0] = deciphered[0] ^ (unsigned int)1;
    decrypted[1] = deciphered[1] ^ (unsigned int)2;
    int i = tx; // 2;
    
    deciphered[0] = encrypted[i];
    deciphered[1] = encrypted[i+1];
    decipher(32, deciphered, (unsigned int*)key);
    decrypted[i] = deciphered[0] ^ encrypted[i-2];
    decrypted[i+1] = deciphered[1] ^ encrypted[i-1];
    i += 2;
}   


__device__ void decipher(unsigned int num_rounds, unsigned int v[2], unsigned int const key[4])
{
    /*
    num_rounds -- the number of iterations in the algorithm, 32 is reccomended
    input_data -- the input data to use, 32 bits of the first 2 elements are used
    key -- 128-bit key to use
    */
    unsigned int i;
    unsigned int v0=v[0], v1=v[1], delta=0x9E3779B9, sum=delta*num_rounds;

    for (i=0; i < num_rounds; i++) {
        v1 -= (((v0 << 4) ^ (v0 >> 5)) + v0) ^ (sum + key[(sum>>11) & 3]);
        sum -= delta;
        v0 -= (((v1 << 4) ^ (v1 >> 5)) + v1) ^ (sum + key[sum & 3]);
    }
    v[0]=v0; v[1]=v1;
}