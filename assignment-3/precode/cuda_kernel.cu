#include "hip/hip_runtime.h"
__device__ void decipher(num_rounds, input_data, key);

__global__ char decrypt_bytes(bytes_in, key) {
	//Gett thread ID
	const int tx = threadIdx.x + (blockIdx.x * blockDim.x);

	//Do the calculation
	//Use tx to something!?!?!
	//something = decipher(num_rounds, input_data, key);
}

__device__ char decipher(num_rounds, input_data, key) {
    /*XTEA implementation in python, decryption.

    Modified version from Simon Biewald (http://varbin.github.io/xtea/)

    Arguments:
    num_rounds -- the number of iterations in the algorithm, 32 is reccomended
    input_data -- the input data to use, 32 bits of the first 2 elements are used
    key -- 128-bit key to use

    returns -- a numpy array containing the deciphered data


    dtype = 32-bit big-endian integer
    */
    uint32_t arrayz[a][b];
    char v0 = input_data[0];
    char v1 = input_data[1];
    char delta = 0x9e3779b9L;
    char mask = 0xffffffffL;
    char sum = (delta*num_rounds) & mask;
    int rounds;
    // ^  	bitwise XOR (eXclusive OR)
    for (rounds = 0; rounds < num_rounds; rounds++)
    {
        v1[rounds] = (v1 - (((v0<<4 ^ v0>>5) + v0) ^ (sum + key[sum>>11 & 3]))) & mask;
        sum = (sum - delta) & mask;
        v0 = (v0 - (((v1<<4 ^ v1>>5) + v1) ^ (sum + key[sum & 3]))) & mask;
    }
    return arrayz[v0][v1];
}